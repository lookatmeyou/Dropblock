#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/dropblock_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void DropblockForward(const int n, const Dtype* in,
    const float* mask, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * mask[index] * scale;
  }
}

__global__ void DropblockExpandMaskGPU(const int n, float* tmp_mask, float* mask,
  float gamma, int block_size, int batch_size, int channels, int feat_size) {
  const int tborder_size = feat_size - block_size + 1;
  // const int tcount = block_size * block_size;
  const int ds1 = channels * tborder_size * tborder_size;
  const int ds2 = tborder_size * tborder_size;
  CUDA_KERNEL_LOOP(index, n) {
    if (tmp_mask[index] <= gamma) {
      const int b = index / ds1;
      const int c = (index % ds1) / ds2;
      const int i = ((index % ds1) % ds2) / tborder_size;
      const int j = ((index % ds1) % ds2) % tborder_size;
      const int p = (b * channels + c) * feat_size * feat_size;
      int tp = p + i * feat_size + j;
      for (int y = 0; y < block_size; ++y, tp += feat_size) {
        memset(mask + tp, 0, block_size * sizeof(float));
      }
    }
  }
}

template <typename Dtype>
void DropblockLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    float* mask =
        static_cast<float*>(rand_vec_.mutable_gpu_data());
    float* tmp_mask =
        static_cast<float*>(tmp_rand_vec_.mutable_gpu_data());
    Dtype t_keep_prob = 1.0 - (1.0 - keep_prob_) * Caffe::get_current_iter() / Caffe::get_max_iter();
    Dtype gamma_ = (1.0 - t_keep_prob) * feat_size_ * feat_size_ /
        (block_size_ * block_size_ * (feat_size_ - block_size_ + 1) * (feat_size_ - block_size_ + 1));
    const int tcount = tmp_rand_vec_.count();
    caffe_gpu_rng_uniform(tcount, 0.0f, 1.0f, tmp_mask);

    caffe_gpu_set<float>(count, 1, mask);
    DropblockExpandMaskGPU<<<CAFFE_GET_BLOCKS(tcount), CAFFE_CUDA_NUM_THREADS>>>(
      tcount, tmp_mask, mask, gamma_, block_size_,
      bottom[0]->shape(0), bottom[0]->shape(1), feat_size_
    );

    scale_ = 1.0 * count / rand_vec_.asum_data();
    DropblockForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, mask, scale_, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void DropblockBackward(const int n, const Dtype* in_diff,
    const float* mask, const float scale, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * mask[index];
  }
}

template <typename Dtype>
void DropblockLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
      const float* mask =
          static_cast<const float*>(rand_vec_.gpu_data());
      const int count = bottom[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropblockBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
          count, top_diff, mask, scale_, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropblockLayer);

}  // namespace caffe
